#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::endl;

#include "io/python/io_python.hpp"
#include "model/model.hpp"
#include "solver/solver.hpp"

//__global__
//void kernel ()
//{
//    return;
//}


int main ()
{
    cout << "Running test_raytracing..." << endl;

    pc::accelerator::list_accelerators();

//    kernel<<<1,1>>>();
    IoPython io ("hdf5", "/home/frederik/Magritte_all/Models/Benchmarks/5_GPU_ray_tracer/test_model.hdf5");

    Model model;
    model.read(io);

    Long2 lengths = model.geometry.get_ray_lengths ();

    cout << "Done." << endl;

    return (0);
}