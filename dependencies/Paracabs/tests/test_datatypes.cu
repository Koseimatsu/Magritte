#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::endl;
#include <vector>
using std::vector;

#include "paracabs.hpp"
namespace pc = paracabs::datatypes;

using Vector3D = pc::Vector3D<double>;

template <typename type>
using Array     = pc::Array <type, pc::MemTypeDefault>;
template <typename type>
using Array_acc = pc::Array <type, pc::MemTypeAccelerator>;



struct Model
{
    size_t length;
    pc::Vector <double> d;

    Model (const size_t s) : length(s), d(s, 8.44) {}

//    Model (const Model& m)
//    {
//        length = m.length;
//        d      = pc::Vector<double> (m.d);
//    }

    inline void f ()
    {
        for (int i = 0; i < length; i++)
        {
            d[i] = 2.29;
        }
    }

    inline void p ()
    {
        for (int i = 0; i < length; i++)
        {
            cout << d[i] << endl;
        }
    }
};


struct Test
{
    size_t length;
    pc::Vector <double> a;

    Model model;


    Test (const size_t s) : length (s), model(3) {a.resize(s);}

//    double* ptr;
//    double* ptr_host;
//    double* ptr_device;
//
//    accel inline type  operator[] (const size_t id) const {return ptr[id];}
//    accel inline type &operator[] (const size_t id)       {return ptr[id];}
//
//    Test (const Test& test)
//    {
//        if (accelerator_code) ptr = ptr_host;
//        else                  ptr = ptr_device;
//    }


    void test ()
    {
        for (int i = 0; i < length; i++)
        {
            a[i] = 3.14;
            cout << a[i] << endl;
        }

        a.copy_vec_to_ptr ();

        accelerated_for (i, length, 1, 1,
        {
            a[i] -= 3;
        })

        a.copy_ptr_to_vec ();

        cout << endl;

        for (int i = 0; i < length; i++)
        {
            cout << a[i] << endl;
        }
    }

    void test2 ()
    {
        model.p();

//        a.copy_vec_to_ptr ();

        accelerated_for (i, model.length, 1, 1,
        {
            model.d[i] -= 3;
        })

        model.d.copy_ptr_to_vec ();

        model.p();
    }

};


//__global__ void addKernel ()
//{
//
//}


int main ()
{
    cout << "Paracabs test datatypes." << endl;

//    Vector3D v1 (1.0, 2.0, 3.0);
//    Vector3D v2 (4.0, 5.0, 6.0);
//    Vector3D v3 = v1 + v2;
//
//    v1.print();
//    v2.print();
//    v3.print();
//
//    cout << v1.dot(v2) << endl;
//
//
//    v1 += v2;
//
//    v1.print();
//    v2.print();
//
//
//    Vector3D v4 = 3.14;
//
//    v4.print();
//
//    (v4 + 1).print();
//
//    v4.print();
//
//    v4 = 7.12;
//
//    v4.print();
//
//
//    const size_t size = 10;

//    array1d <vector3d <double>, MemTypeDefault>     arr1 (size);
//    array1d <vector3d <double>, MemTypeAccelerator> arr2 (size);
//
//    for (size_t i = 0; i < size; i++)
//    {
//        arr1[i] = 1.0;
//    }
//
//    for (size_t i = 0; i < size; i++)
//    {
//        arr1[i].print();
//    }
//
//    for (size_t i = 0; i < size; i++)
//    {
//        arr2[i] = 1.0;
//    }
//
//    for (size_t i = 0; i < size; i++)
//    {
//        arr2[i].print();
//    }

    Test t (10);

//    t.test();

//    size_t length = 15;
//    pc::Vector <double> a (length);
//
//    for (int i = 0; i < length; i++)
//    {
//        a[i] = 3.14;
//        cout << a[i] << endl;
//    }
//
//    a.copy_vec_to_ptr ();
//
//    accelerated_for_outside_class (i, length, 1, 1,
//    {
//        a[i] -= 3;
//    })
//
//    a.copy_ptr_to_vec ();
//
//    cout << endl;
//
//    for (int i = 0; i < length; i++)
//    {
//        cout << a[i] << endl;
//    }

    cout << "Done." << endl;

//    t.model.p();
//    t.model.f();
//    t.model.p();

    t.test2();



    return (0);
}